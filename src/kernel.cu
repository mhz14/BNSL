#include "hip/hip_runtime.h"
#include "BNSL_GPU.cuh"

int * valuesRange;

int nodesNum = 0;

int * samplesValues;

int samplesNum;

int allParentSetNumPerNode;

double * dev_lsTable;

int* globalBestGraph;
int* topSort;
double globalBestScore;

int begin = 0;

int main() {
	calcCPUTimeStart("init.");
	BNSL_init();
	calcCPUTimeEnd();

	calcCPUTimeStart("calcLS.");
	BNSL_calcLocalScore();
	calcCPUTimeEnd();

	calcCPUTimeStart("start.");
	BNSL_start();
	calcCPUTimeEnd();

	printf("Bayesian Network learned:\n");
	for (int i = 0; i < nodesNum; i++) {
		for (int j = 0; j < nodesNum; j++) {
			printf("%d ", globalBestGraph[i * nodesNum + j]);
		}
		printf("\n");
	}
	printf("Best Score: %f \n", globalBestScore);
	printf("Best Topology: ");
	for (int i = 0; i < nodesNum; i++) {
		printf("%d ", topSort[i]);
	}
	printf("\n");

	calcCPUTimeStart("finish.");
	BNSL_finish();
	calcCPUTimeEnd();

	return 0;
}

void CheckCudaError(hipError_t err, char const* errMsg) {
	if (err == hipSuccess)
		return;
	printf("%s\nError Message: %s.\n", errMsg, hipGetErrorString(err));
	exit(EXIT_FAILURE);
}

void calcCPUTimeStart(char const *message) {
	begin = clock();
	printf("%s", message);
}

void calcCPUTimeEnd() {
	printf("Elapsed CPU time is %dms\n", (clock() - begin) / 1000);
}

void BNSL_init() {
	readNodeInfo();
	readSamples();
}

void BNSL_calcLocalScore() {

	int i;
	allParentSetNumPerNode = 0;
	for (i = 0; i <= CONSTRAINTS; i++) {
		allParentSetNumPerNode = allParentSetNumPerNode + C(i, nodesNum - 1);
	}

	int* dev_valuesRange;
	int* dev_samplesValues;
	int* dev_N;

	// calculate max different values number for all pair of child and parent set
	int valuesMaxNum = calcValuesMaxNum();

	// malloc in GPU global mem.
	CUDA_CHECK_RETURN(
			hipMalloc(&dev_lsTable,
					nodesNum * allParentSetNumPerNode * sizeof(double)),
			"dev_lsTable hipMalloc failed.");
	CUDA_CHECK_RETURN(hipMalloc(&dev_valuesRange, nodesNum * sizeof(int)),
			"dev_valuesRange hipMalloc failed.");
	CUDA_CHECK_RETURN(
			hipMalloc(&dev_samplesValues, samplesNum * nodesNum * sizeof(int)),
			"dev_samplesValues hipMalloc failed.");
	CUDA_CHECK_RETURN(
			hipMalloc(&dev_N,
					allParentSetNumPerNode * valuesMaxNum * sizeof(int)),
			"dev_N hipMalloc failed.");

	// copy data from CPU mem to GPU mem.
	CUDA_CHECK_RETURN(
			hipMemcpy(dev_valuesRange, valuesRange, nodesNum * sizeof(int),
					hipMemcpyHostToDevice),
			"valuesRange -> dev_valuesRange failed.");
	CUDA_CHECK_RETURN(
			hipMemcpy(dev_samplesValues, samplesValues,
					samplesNum * nodesNum * sizeof(int),
					hipMemcpyHostToDevice),
			"samplesValues -> dev_samplesValues failed.");
	CUDA_CHECK_RETURN(
			hipMemset(dev_N, 0,
					allParentSetNumPerNode * valuesMaxNum * sizeof(int)),
			"dev_N hipMemset failed.");

	int blockNum = (allParentSetNumPerNode + 1) / 256 + 1;
	calAllLocalScore_kernel<<<blockNum, 256>>>(dev_valuesRange,
			dev_samplesValues, dev_N, dev_lsTable, samplesNum, nodesNum,
			allParentSetNumPerNode, valuesMaxNum);
	CUDA_CHECK_RETURN(hipDeviceSynchronize(),
			"calAllLocalScore_kernel failed on running.");

	CUDA_CHECK_RETURN(hipFree(dev_valuesRange),
			"dev_valuesRange hipFree failed.");
	CUDA_CHECK_RETURN(hipFree(dev_samplesValues),
			"dev_samplesValues hipFree failed.");
	CUDA_CHECK_RETURN(hipFree(dev_N), "dev_N hipFree failed.");

	free(valuesRange);
	free(samplesValues);
}

void BNSL_start() {
	int i, j, iter, offset;
	double oldScore = -DBL_MAX, newScore = 0.0;
	globalBestScore = -DBL_MAX;
	globalBestGraph = (int *) malloc(sizeof(int) * nodesNum * nodesNum);
	topSort = (int *) malloc(sizeof(int) * nodesNum);

	double * nodeScore = (double *) malloc(nodesNum * sizeof(double));
	int * bestParentSet = (int *) malloc(
			(CONSTRAINTS + 1) * nodesNum * sizeof(int));

	int * dev_order;
	int * dev_bestParentSet;
	double * dev_nodeScore;

	CUDA_CHECK_RETURN(hipMalloc(&dev_order, nodesNum * sizeof(int)),
			"dev_order hipMalloc failed.");
	CUDA_CHECK_RETURN(hipMalloc(&dev_nodeScore, nodesNum * sizeof(double)),
			"dev_nodeScore hipMalloc failed.");
	CUDA_CHECK_RETURN(
			hipMalloc(&dev_bestParentSet, nodesNum * (CONSTRAINTS + 1) * sizeof(int)),
			"dev_bestParentSet hipMalloc failed.");

	int * newOrder = (int *) malloc(sizeof(int) * nodesNum);
	randInitOrder(newOrder);
	int * oldOrder = (int *) malloc(sizeof(int) * nodesNum);

	int maxIterNum = 1;
	for (iter = 0; iter < maxIterNum; iter++) {

		//randSwapTwoNode(newOrder);
		newOrder[0] = 1;
		newOrder[1] = 2;
		newOrder[2] = 3;
		newOrder[3] = 4;
		newOrder[4] = 5;

		CUDA_CHECK_RETURN(
				hipMemcpy(dev_order, newOrder, nodesNum * sizeof(int),
						hipMemcpyHostToDevice),
				"newOrder -> dev_order failed.");

		// use GPU to calculate order score
		calOrderScore_kernel<<<nodesNum, 256, 256 * 8>>>(dev_lsTable, dev_order,
				dev_nodeScore, dev_bestParentSet, allParentSetNumPerNode,
				nodesNum);

		CUDA_CHECK_RETURN(
				hipMemcpy(nodeScore, dev_nodeScore, nodesNum * sizeof(double),
						hipMemcpyDeviceToHost),
				"dev_nodeScore -> nodeScore failed.");
		CUDA_CHECK_RETURN(
				hipMemcpy(bestParentSet, dev_bestParentSet, nodesNum * (CONSTRAINTS + 1) * sizeof(int), hipMemcpyDeviceToHost),
				"dev_bestParentSet -> bestParentSet failed.");

		newScore = 0.0;
		for (i = 0; i < nodesNum; i++) {
			newScore += nodeScore[i];
		}

		// use Metropolis-Hastings rule
		srand((unsigned int) time(NULL));
		double u = rand() / (double) RAND_MAX;
		if (log(u) < newScore - oldScore) {
			oldScore = newScore;
			memcpy(oldOrder, newOrder, nodesNum * sizeof(int));
		}

		// search for best graph
		if (newScore > globalBestScore) {
			globalBestScore = newScore;
			memset(globalBestGraph, 0, nodesNum * nodesNum * sizeof(int));

			for (i = 0; i < nodesNum; i++) {
				for (j = 1, offset = i * (CONSTRAINTS + 1);
						j <= bestParentSet[offset]; j++) {
					globalBestGraph[(bestParentSet[offset + j] - 1) * nodesNum
							+ i] = 1;
				}
			}

			memcpy(topSort, newOrder, nodesNum * sizeof(int));
		}
	}

	CUDA_CHECK_RETURN(hipFree(dev_lsTable), "dev_lsTable hipFree failed.");
	CUDA_CHECK_RETURN(hipFree(dev_order), "dev_order hipFree failed.");
	CUDA_CHECK_RETURN(hipFree(dev_nodeScore),
			"dev_nodeScore hipFree failed.");
	CUDA_CHECK_RETURN(hipFree(dev_bestParentSet),
			"dev_bestParentSet hipFree failed.");
	free(nodeScore);
	free(newOrder);
	free(oldOrder);
	free(bestParentSet);
}

void BNSL_finish() {
	free(topSort);
	free(globalBestGraph);
}

void readNodeInfo() {
	FILE * inFile = fopen(NODEINFO_PATH, "r");

	char cur = fgetc(inFile);
	while (cur != EOF) {
		if (cur == '\n')
			nodesNum++;
		cur = fgetc(inFile);
	}
	nodesNum++;

	rewind(inFile);
	valuesRange = (int *) malloc(sizeof(int) * nodesNum);
	int i;
	for (i = 0; i < nodesNum; i++) {
		fscanf(inFile, "%d", &(valuesRange[i]));
	}

	fclose(inFile);

}

void readSamples() {
	FILE * inFile = fopen(SAMPLES_PATH, "r");
	int i, j, value;

	samplesNum = 0;
	char cur = fgetc(inFile);
	while (cur != EOF) {
		if (cur == '\n')
			samplesNum++;
		cur = fgetc(inFile);
	}
	samplesNum++;

	samplesValues = (int *) malloc(sizeof(int) * samplesNum * nodesNum);
	rewind(inFile);
	for (i = 0; i < samplesNum; i++) {
		for (j = 0; j < nodesNum; j++) {
			fscanf(inFile, "%d", &value);
			samplesValues[i * nodesNum + j] = value - 1;
		}
	}

	fclose(inFile);
}

int compare(const void*a, const void*b) {
	return *(int*) a - *(int*) b;
}

long C(int n, int m) {

	if (n > m || n < 0 || m < 0)
		return -1;

	int k, res = 1;
	for (k = 1; k <= n; k++) {
		res = (res * (m - n + k)) / k;
	}
	return res;
}

void randInitOrder(int * s) {
	for (int i = 0; i < nodesNum; i++) {
		s[i] = i + 1;
	}
	int swap, r;
	srand((unsigned int) time(NULL));
	for (int i = nodesNum - 1; i > 0; i--) {
		r = rand() % i;
		swap = s[r];
		s[r] = s[i];
		s[i] = swap;
	}
}

void selectTwoNodeToSwap(int *n1, int *n2) {
	*n1 = rand() % nodesNum;
	*n2 = rand() % nodesNum;
	if (*n1 == *n2) {
		*n2 = rand() % (nodesNum - 1);
		if (*n2 >= *n1) {
			*n2++;
		}
	}
}

void randSwapTwoNode(int *order) {
	int n1 = 0, n2 = 0, temp;
	selectTwoNodeToSwap(&n1, &n2);
	temp = order[n1];
	order[n1] = order[n2];
	order[n2] = temp;
}

int calcValuesMaxNum() {
	int * valuesRangeToSort = (int *) malloc(nodesNum * sizeof(int));
	memcpy(valuesRangeToSort, valuesRange, nodesNum * sizeof(int));
	qsort(valuesRangeToSort, nodesNum, sizeof(int), compare);
	int valuesMaxNum = 1;
	for (int i = nodesNum - CONSTRAINTS - 1; i < nodesNum; i++) {
		valuesMaxNum *= valuesRangeToSort[i];
	}
	free(valuesRangeToSort);
	return valuesMaxNum;
}

__global__ void calOrderScore_kernel(double * dev_lsTable, int * dev_order,
		double * dev_nodeScore, int * dev_bestParentSet, int allParentSetNumPerNode,
		int nodesNum) {

	int parentSetNumInOrder = 0;
	int i, s;
	int curPos = blockIdx.x;
	int curNode = dev_order[curPos];
	for (i = 0; i <= CONSTRAINTS && i < curPos + 1; i++) {
		parentSetNumInOrder += C_kernel(i, curPos);
	}

	extern __shared__ double result[];
	result[threadIdx.x] = -DBL_MAX;
	__syncthreads();
	int combi[CONSTRAINTS];
	int size = 0;
	if (threadIdx.x < parentSetNumInOrder) {
		findComb_kernel(curPos + 1, threadIdx.x, &size, combi);

		int parentSet[CONSTRAINTS];
		for (i = 0; i < size; i++) {
			parentSet[i] = dev_order[combi[i] - 1];
		}

		sortArray_kernel(parentSet, size);

		for (i = 0; i < size; i++) {
			if (parentSet[i] > curNode) {
				parentSet[i] -= 1;
			}
		}

		int index = 0;
		if (size > 0) {
			index = findIndex_kernel(size, parentSet, nodesNum);
		}

		result[threadIdx.x] = dev_lsTable[(curNode - 1) * allParentSetNumPerNode + index];
	}

	__syncthreads();

	s = blockDim.x / 2;
	if (threadIdx.x < s) {
		if (result[threadIdx.x] >= result[threadIdx.x + s]) {
			result[threadIdx.x + s] = threadIdx.x;
		} else {
			result[threadIdx.x] = result[threadIdx.x + s];
			result[threadIdx.x + s] = threadIdx.x + s;
		}
	}

	__syncthreads();

	for (s = blockDim.x / 4; s > 0; s >>= 1) {
		if (threadIdx.x < s) {
			if (result[threadIdx.x] >= result[threadIdx.x + s]) {
				result[threadIdx.x + s] = result[threadIdx.x + 2 * s];
			} else {
				result[threadIdx.x] = result[threadIdx.x + s];
				result[threadIdx.x + s] = result[threadIdx.x + 3 * s];
			}
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		dev_nodeScore[curPos] = result[0];
	}

	if (threadIdx.x == result[1]) {
		dev_bestParentSet[(curNode - 1) * (CONSTRAINTS + 1)] = size;
		for (i = 0; i < size; i++) {
			dev_bestParentSet[(curNode - 1) * (CONSTRAINTS + 1) + i + 1] =
					dev_order[combi[i] - 1];
		}
	}
}

__device__ double calLocalScore_kernel(int *dev_valuesRange,
		int *dev_samplesValues, int *dev_N, int samplesNum, int size,
		int* parentSet, int curNode, int nodesNum, int valuesMaxNum) {

	int curNodeValuesNum = dev_valuesRange[curNode];
	int valuesNum = 1;
	int i, j;
	for (i = 0; i < size; i++) {
		valuesNum = valuesNum * dev_valuesRange[parentSet[i] - 1];
	}

	int *N = dev_N + (blockIdx.x * blockDim.x + threadIdx.x) * valuesMaxNum;
	for (i = 0; i < valuesMaxNum; i++) {
		N[i] = 0;
	}
	int pvalueIndex = 0;
	for (i = 0; i < samplesNum; i++) {
		pvalueIndex = 0;
		for (j = 0; j < size; j++) {
			pvalueIndex = pvalueIndex * dev_valuesRange[parentSet[j] - 1]
					+ dev_samplesValues[i * nodesNum + parentSet[j] - 1];
		}

		N[pvalueIndex * curNodeValuesNum
				+ dev_samplesValues[i * nodesNum + curNode]]++;
	}

	double alpha = ALPHA / (curNodeValuesNum * valuesNum);
	double localScore = size * log(GAMMA);
	for (i = 0; i < valuesNum; i++) {
		int sum = 0;
		for (j = 0; j < curNodeValuesNum; j++) {
			int cur = i * curNodeValuesNum + j;
			if (N[cur] != 0) {
				localScore = localScore + lgamma(N[cur] + alpha)
						- lgamma(alpha);
				sum = sum + N[cur];
			}
		}
		localScore = localScore + lgamma(alpha * curNodeValuesNum)
				- lgamma(alpha * curNodeValuesNum + sum);
	}

	return localScore;
}

__global__ void calAllLocalScore_kernel(int *dev_valuesRange,
		int *dev_samplesValues, int *dev_N, double *dev_lsTable, int samplesNum,
		int nodesNum, int allParentSetNumPerNode, int valuesMaxNum) {

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < allParentSetNumPerNode) {
		int size = 0;
		int combination[CONSTRAINTS], parentSet[CONSTRAINTS];
		findComb_kernel(nodesNum, id, &size, combination);
		int i, curNode;
		for (curNode = 0; curNode < nodesNum; curNode++) {
			for (i = 0; i < size; i++) {
				parentSet[i] = combination[i];
			}
			recoverComb_kernel(curNode, parentSet, size);
			double result = calLocalScore_kernel(dev_valuesRange,
					dev_samplesValues, dev_N, samplesNum, size, parentSet,
					curNode, nodesNum, valuesMaxNum);
			dev_lsTable[curNode * allParentSetNumPerNode + id] = result;
		}
	}
}

__device__ long C_kernel(int n, int m) {

	if (n > m || n < 0 || m < 0)
		return -1;

	int k, res = 1;
	for (k = 1; k <= n; k++) {
		res = (res * (m - n + k)) / k;
	}
	return res;
}

__device__ void recoverComb_kernel(int curNode, int* combi, int size) {

	for (int i = 0; i < size; i++) {
		if (combi[i] >= curNode + 1) {
			combi[i] = combi[i] + 1;
		}
	}
}

__device__ void findComb_kernel(int nodesNum, int index, int* size,
		int* combi) {

	if (index == 0) {
		*size = 0;
	} else {
		int k = 1;
		int limit = C_kernel(k, nodesNum - 1);
		while (index > limit) {
			k++;
			limit = limit + C_kernel(k, nodesNum - 1);
		}
		index = index - limit + C_kernel(k, nodesNum - 1);
		*size = k;

		int base = 0;
		int n = nodesNum - 1;
		int i, sum, shift;
		int sum_new = 0;

		for (i = 1; i < k; i++) {
			sum = 0;
			for (shift = 1; shift <= n; shift++) {
				sum_new = sum + C_kernel(k - i, n - shift);
				if (sum_new < index) {
					sum = sum_new;
				} else {
					break;
				}
			}
			combi[i - 1] = base + shift;
			n = n - shift;
			index = index - sum;
			base = combi[i - 1];
		}
		combi[k - 1] = base + index;
	}
}

__device__ void sortArray_kernel(int * s, int n) {
	int min;
	int id = -1;
	for (int i = 0; i < n - 1; i++) {
		min = s[i];
		id = i;
		for (int j = i + 1; j < n; j++) {
			if (s[j] < min) {
				min = s[j];
				id = j;
			}
		}
		if (i != id) {
			int swap = s[i];
			s[i] = s[id];
			s[id] = swap;
		}
	}
}

__device__ int findIndex_kernel(int k, int* combi, int nodesNum) {
	int index = 1;
	int i, j;
	int * newCombi = (int *) malloc(sizeof(int) * (k + 1));
	newCombi[0] = 0;
	for (i = 1; i <= k; i++) {
		newCombi[i] = combi[i - 1];
	}
	for (i = 1; i <= k; i++) {
		for (j = newCombi[i - 1] + 1; j < newCombi[i]; j++) {
			index = index + C_kernel(k - i, nodesNum - 1 - j);
		}
	}

	free(newCombi);

	for (i = 1; i < k; i++) {
		index = index + C_kernel(i, nodesNum - 1);
	}

	return index;
}
